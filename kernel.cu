#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// General macros
#define DEFR 0.73f // default colors (chartruese)
#define DEFG 0.65f
#define DEFB 0.06f

// TARGA macros
#define IDLENGTH (char)31
#define IDMSG "jefff-generated raytraced image"
#define CMT (char)0
#define ITC (char)2
#define XORIGIN (short)0
#define YORIGIN (short)0
#define IPS (char)24
#define IDB (char)0b00010000

typedef struct {
    float* r; // individual channels
    float* b;
    float* g; 
    unsigned int width;
    unsigned int height;
} IMAGE;

typedef float VECTOR4[4];
typedef float MATRIX4[4][4];

/* Generic CUDA setup/run
* IMAGE* canvas: image to give to the gpu
* Will make a UV display on the canvas (right now)
*/
unsigned int cudaSetup(IMAGE* canvas);

/* Writes a list of pixels to a TGA
* const char* fn: c-string of the filename
* const PIXEL* pixels: array of pixels to write
* const unsigned int pixel_len: number of pixels (prevent overflows)
* Writes a type 2 TARGA 24 file
*/
unsigned int writeTGA(const char* fn, IMAGE* pixels);

__global__ void uvFill(float* r, float* g, float* b, unsigned int* width, unsigned int* height) {
    unsigned int index = blockIdx.x + (blockIdx.y * width[0]); // terrible, i know
    r[index] = (float)blockIdx.x / (float)width[0]; 
    g[index] = (float)blockIdx.y / (float)height[0];
    b[index] = 0.5f;
}

int main(int argc, char* argv[]) {
    bool err = false;

    printf("Running with args: "); // handling the filename and args
    for (unsigned int i = 0; i < argc; i++) { printf(argv[i]); printf(" "); }
    printf(" \n");
    if (argc <= 1) {
        printf("Must provide filename as argument! \n(If you did provide a filename as an argument, simply move the name such that it is the second argument. This is due to the fact some systems provide the command path as the first argument, whereas some may not.) \n");
        err = true;
        goto MAINERR;
    }

    IMAGE image{};
    image.width = 1280; image.height = 720; // 720p

    image.r = (float*)malloc(sizeof(float) * (image.width * image.height));
    image.g = (float*)malloc(sizeof(float) * (image.width * image.height));
    image.b = (float*)malloc(sizeof(float) * (image.width * image.height));

    if (image.r == NULL || image.g == NULL || image.b == NULL) {
        printf("NULL pointer when allocating image! r: %p g: %p b: %p\n", image.r, image.g, image.b);
        err = true;
        goto MAINERR;
    }

    for (unsigned int i = 0; i < image.width * image.height; i++) {
        image.r[i] = DEFR;
        image.g[i] = DEFG;
        image.b[i] = DEFB;
    }

    switch (cudaSetup(&image)) {
        case 0: break;
        default: printf("Error in cudaSetup()\n");  err = true; break;
    }

    if(err) goto MAINERR;

    switch (writeTGA(argv[1], &image)) {
        case 0: printf("TGA written succesfully! \n"); break;
        case 1: printf("TGA file couldn't be opened/created! \n"); err = true; break;
        default: printf("Unknown error when writing TGA! \n"); err = true; break;
    }

    if(err) goto MAINERR;

    MAINERR:
    free(image.r);
    free(image.g);
    free(image.b);
    if (err) {
        printf("Returning with an error!\n");
        return -1;
    }

    return 0;
}

unsigned int cudaSetup(IMAGE* canvas) {
    dim3 blockSize(1, 1);
    dim3 numBlocks(canvas->width, canvas->height);

    printf("Running with:\nWidth/Height: %i * %i\nBlock Size: %i * %i\nBlock Grid: %i * %i\n", canvas->width, canvas->height, blockSize.x, blockSize.y, numBlocks.x, numBlocks.y);

    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("Couldn't set device!\n");
        goto SETUPERR;
    }

    float* device_r;
    float* device_g;
    float* device_b;
    unsigned int* device_wi;
    unsigned int* device_hi;

    err = hipMalloc((void**)&device_r, canvas->width * canvas->height * sizeof(float));
    err = hipMalloc((void**)&device_g, canvas->width * canvas->height * sizeof(float));
    err = hipMalloc((void**)&device_b, canvas->width * canvas->height * sizeof(float));
    if (err != hipSuccess) {
        printf("Couldn't hipMalloc() the rgb channels!\n");
        goto SETUPERR;
    }

    err = hipMalloc((void**)&device_wi, sizeof(float));
    err = hipMalloc((void**)&device_hi, sizeof(float));
    if (err != hipSuccess) {
        printf("Couldn't hipMalloc() the width/height!\n");
        goto SETUPERR;
    }

    err = hipMemcpy(device_r, canvas->r, canvas->width * canvas->height * sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(device_g, canvas->g, canvas->width * canvas->height * sizeof(float), hipMemcpyHostToDevice);
    err = hipMemcpy(device_b, canvas->b, canvas->width * canvas->height * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Couldn't hipMemcpy() the rgb channels! (host >> device)\n");
        goto SETUPERR;
    }

    err = hipMemcpy(device_wi, &canvas->width, sizeof(unsigned int), hipMemcpyHostToDevice);
    err = hipMemcpy(device_hi, &canvas->height, sizeof(unsigned int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Couldn't hipMemcpy() the width/height! (host >> device)\n");
        goto SETUPERR;
    }

    uvFill<<<blockSize, numBlocks>>>(device_r, device_g, device_b, device_wi, device_hi);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in kernel: %s, %s\n", hipGetErrorName(err), hipGetErrorString(err));
        goto SETUPERR;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Couldn't sync threads!\n");
        goto SETUPERR;
    }

    err = hipMemcpy(canvas->r, device_r, canvas->width * canvas->height * sizeof(float), hipMemcpyDeviceToHost);
    err = hipMemcpy(canvas->g, device_g, canvas->width * canvas->height * sizeof(float), hipMemcpyDeviceToHost);
    err = hipMemcpy(canvas->b, device_b, canvas->width * canvas->height * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Couldn't hipMemcpy() the rgb channels! (device >> host)\n");
        goto SETUPERR;
    }

    SETUPERR:
    err = hipFree(device_r);
    err = hipFree(device_g);
    err = hipFree(device_b);
    err = hipFree(device_wi);
    err = hipFree(device_hi);
    if (err != hipSuccess) return -1;
    return 0;
}

unsigned int writeTGA(const char* fn, IMAGE* pixels) {
    FILE* targa = fopen(fn, "wb");
    if (targa == NULL) {
        return 1;
    }
    fputc(IDLENGTH, targa); // length of identification msg
    fputc(CMT, targa); // color map type (0 = ignore)
    fputc(ITC, targa); // type of targa (2)
    for (unsigned int i = 0; i < 5; i++) { // generates blank area where the color map is, since it should be ignored
        fputc(0, targa);
    }
    fputc((XORIGIN & 0x00FF), targa); fputc((XORIGIN & 0xFF00) / 256, targa); // x and y origins, weird bitmapping is to concatenate the usually 2-byte short into a 1-byte char
    fputc((YORIGIN & 0x00FF), targa); fputc((YORIGIN & 0xFF00) / 256, targa);
    fputc((pixels->width & 0x00FF), targa); fputc((pixels->width & 0xFF00) / 256, targa); // width and height using same bitmapping technique
    fputc((pixels->height & 0x00FF), targa); fputc((pixels->height & 0xFF00) / 256, targa);
    fputc(IPS, targa);
    fputc(IDB, targa);
    const char* idmsg = IDMSG;
    for (unsigned int i = 0; i < IDLENGTH; i++) {
        fputc(idmsg[i], targa);
    }
    for (unsigned int i = 0; i < pixels->height * pixels->width; i++) {
        fputc((int)roundf(pixels->b[i] * 255), targa);
        fputc((int)roundf(pixels->g[i] * 255), targa);
        fputc((int)roundf(pixels->r[i] * 255), targa);
    }
    fclose(targa);
    return 0;
}