﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define WIDTH (short)1
#define HEIGHT (short)1

// TARGA macros
#define VERSION (char)0
#define IDLENGTH (char)31
#define IDMSG "jefff-generated raytraced image"
#define CMT (char)0
#define ITC (char)2
#define XORIGIN (short)0
#define YORIGIN (short)0
#define IPS (char)24
#define IDB (char)0b00010000

typedef struct {
    float r;
    float g;
    float b;
} PIXEL;

/* Writes a list of pixels to a TGA
* const char* fn: c-string of the filename
* const PIXEL* pixels: array of pixels to write
* const unsigned int pixel_len: number of pixels (prevent overflows)
* Writes a type 2 TARGA 24 file
*/
unsigned int writeTGA(const char* fn, const PIXEL* pixels, const unsigned int pixel_len);

int main(int argc, char* argv[])
{
    printf("Running with args: ");
    for (unsigned int i = 0; i < argc; i++) { printf(argv[i]); printf(argv[i]); }
    printf(" \n");
    if (argc <= 1) {
        printf("Must provide filename as argument! \n(If you did provide a filename as an argument, simply move the name such that it is the second argument. This is due to the fact some systems provide the command path as the first argument, whereas some may not.) \n");
        exit(-1);
    }
    const unsigned int pixel_len = WIDTH * HEIGHT;
    PIXEL image[pixel_len] = {{1.0f, 0.7f, 0.9f}};

    switch (writeTGA(argv[1], image, pixel_len)) {
        case 0: printf("TGA written succesfully! \n"); break;
        case 1: printf("TGA file couldn't be opened/created! \n"); break;
        default: printf("Unknown error when writing TGA! \n"); break;
    }

    return 0;
}

unsigned int writeTGA(const char* fn, const PIXEL* pixels, const unsigned int pixel_len) {
    FILE* targa = fopen(fn, "wb");
    if (targa == NULL) {
        return 1;
    }
    fputc(IDLENGTH, targa); // length of identification msg
    fputc(CMT, targa); // color map type (0 = ignore)
    fputc(ITC, targa); // type of targa (2)
    for (unsigned int i = 0; i < 5; i++) { // generates blank area where the color map is, since it should be ignored
        fputc(0, targa);
    }
    fputc((XORIGIN & 0x00FF), targa); fputc((XORIGIN & 0xFF00) / 256, targa); // x and y origins, weird bitmapping is to concatenate the usually 2-byte short into a 1-byte char
    fputc((YORIGIN & 0x00FF), targa); fputc((YORIGIN & 0xFF00) / 256, targa);
    fputc((WIDTH & 0x00FF), targa); fputc((WIDTH & 0xFF00) / 256, targa); // width and height using same bitmapping technique
    fputc((HEIGHT & 0x00FF), targa); fputc((HEIGHT & 0xFF00) / 256, targa);
    fputc(IPS, targa);
    fputc(IDB, targa);
    const char* idmsg = IDMSG;
    for (unsigned int i = 0; i < IDLENGTH; i++) {
        fputc(idmsg[i], targa);
    }
    for (unsigned int i = 0; i < pixel_len; i++) {
        fputc((char)roundf(pixels[i].b), targa);
        fputc((char)roundf(pixels[i].g), targa);
        fputc((char)roundf(pixels[i].r), targa);
    }
    return 0;
}